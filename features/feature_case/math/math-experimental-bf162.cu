//====-------- math-experimental-bf162.cu ---------- *- CUDA -* -----------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "hip/hip_bf16.h"

using namespace std;

typedef vector<__hip_bfloat162> bf162_vector;
typedef pair<__hip_bfloat162, int> bf162i_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

void checkResult(const string &FuncName, const vector<float2> &Inputs,
                 const float2 &Expect, const float2 &Result,
                 const int precision) {
  cout << FuncName << "({" << Inputs[0].x << ", " << Inputs[0].y << "}";
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", {" << Inputs[i].x << ", " << Inputs[i].y << "}";
  }
  cout << ") = " << fixed << setprecision(precision) << "{" << Result.x << ", "
       << Result.y << "} (expect {" << Expect.x - pow(10, -precision) << " ~ "
       << Expect.x + pow(10, -precision) << ", "
       << Expect.y - pow(10, -precision) << " ~ "
       << Expect.y + pow(10, -precision) << ")";
  cout.unsetf(ios::fixed);
  check(abs(Result.x - Expect.x) < pow(10, -precision) &&
        abs(Result.y - Expect.y) < pow(10, -precision));
}

void checkResult(const string &FuncName, const vector<__hip_bfloat162> &Inputs,
                 const __hip_bfloat162 &Expect, const float2 &Result,
                 const int precision) {
  vector<float2> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back({__bfloat162float(it.x), __bfloat162float(it.y)});
  }
  float2 FExpect{__bfloat162float(Expect.x), __bfloat162float(Expect.y)};
  checkResult(FuncName, FInputs, FExpect, Result, precision);
}

// Bfloat162 Arithmetic Functions

__global__ void habs2(float *const Result, __hip_bfloat162 Input1) {
  auto ret = __habs2(Input1);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHabs2Cases(
    const vector<pair<__hip_bfloat162, bf162i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    habs2<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__habs2", {TestCase.first}, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
  }
}

__global__ void hfma2(float *const Result, __hip_bfloat162 Input1,
                      __hip_bfloat162 Input2, __hip_bfloat162 Input3) {
  auto ret = __hfma2(Input1, Input2, Input3);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHfma2Cases(const vector<pair<bf162_vector, bf162i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma2<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                    TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma2", TestCase.first, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hfma2_relu(float *const Result, __hip_bfloat162 Input1,
                           __hip_bfloat162 Input2, __hip_bfloat162 Input3) {
  auto ret = __hfma2_relu(Input1, Input2, Input3);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHfma2_reluCases(
    const vector<pair<bf162_vector, bf162i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma2_relu<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                         TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma2_relu", TestCase.first, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

__global__ void hfma2_sat(float *const Result, __hip_bfloat162 Input1,
                          __hip_bfloat162 Input2, __hip_bfloat162 Input3) {
  auto ret = __hfma2_sat(Input1, Input2, Input3);
  Result[0] = __low2float(ret);
  Result[1] = __high2float(ret);
}

void testHfma2_satCases(
    const vector<pair<bf162_vector, bf162i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, 2 * sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    hfma2_sat<<<1, 1>>>(Result, TestCase.first[0], TestCase.first[1],
                        TestCase.first[2]);
    hipDeviceSynchronize();
    checkResult("__hfma2_sat", TestCase.first, TestCase.second.first,
                {Result[0], Result[1]}, TestCase.second.second);
    if (TestCase.first.size() != 3) {
      failed++;
      cout << " ---- failed" << endl;
      return;
    }
  }
}

int main() {
  testHabs2Cases({
      {{-0.3, -5}, {{0.30078125, 5}, 15}},
      {{0.3, 5}, {{0.30078125, 5}, 15}},
      {{0.4, 0.2}, {{0.400390625, 0.2001953125}, 16}},
      {{0.3, 0.5}, {{0.30078125, 0.5}, 16}},
      {{3, 5}, {{3, 5}, 15}},
  });
  testHfma2Cases({
      {{{-0.3, -0.5}, {-0.4, -0.6}, {-0.2, -0.7}},
       {{-0.07958984375, -0.3984375}, 16}},
      {{{0.3, 0.5}, {-0.4, 0.6}, {-0.1, 0.2}}, {{-0.220703125, 0.5}, 16}},
      {{{0.3, 0.5}, {0.4, 0.2}, {0.1, 0.1}}, {{0.220703125, 0.2001953125}, 16}},
      {{{0.3, 0.5}, {0.4, 0.6}, {0, 0.3}}, {{0.12060546875, 0.6015625}, 16}},
      {{{3, 5}, {4, 6}, {5, 8}}, {{17, 38}, 14}},
  });
  testHfma2_reluCases({
      {{{-0.3, -0.5}, {-0.4, -0.6}, {-0.2, -0.7}}, {{0, 0}, 37}},
      {{{0.3, 0.5}, {-0.4, 0.6}, {-0.1, 0.2}}, {{0, 0.5}, 16}},
      {{{0.3, 0.5}, {0.4, 0.2}, {0.1, 0.1}}, {{0.220703125, 0.2001953125}, 16}},
      {{{0.3, 0.5}, {0.4, 0.6}, {0, 0.3}}, {{0.12060546875, 0.6015625}, 16}},
      {{{3, 5}, {4, 6}, {5, 8}}, {{17, 38}, 14}},
  });
  testHfma2_satCases({
      {{{-0.3, -0.5}, {-0.4, -0.6}, {-0.2, -0.7}}, {{0, 0}, 37}},
      {{{0.3, 0.5}, {-0.4, 0.6}, {-0.1, 0.2}}, {{0, 0.5}, 16}},
      {{{0.3, 0.5}, {0.4, 0.2}, {0.1, 0.1}}, {{0.220703125, 0.2001953125}, 16}},
      {{{0.3, 0.5}, {0.4, 0.6}, {0, 0.3}}, {{0.12060546875, 0.6015625}, 16}},
      {{{3, 5}, {4, 6}, {5, 8}}, {{1, 1}, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
