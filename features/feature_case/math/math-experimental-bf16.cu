//====-------- math-experimental-bf16.cu- -------- *- CUDA -* -------------===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "hip/hip_bf16.h"

using namespace std;

typedef pair<__hip_bfloat16, int> bf16i_pair;

int passed = 0;
int failed = 0;

void check(bool IsPassed) {
  if (IsPassed) {
    cout << " ---- passed" << endl;
    passed++;
  } else {
    cout << " ---- failed" << endl;
    failed++;
  }
}

void checkResult(const string &FuncName, const vector<float> &Inputs,
                 const float &Expect, const float &Result,
                 const int precision) {
  cout << FuncName << "(" << Inputs[0];
  for (size_t i = 1; i < Inputs.size(); ++i) {
    cout << ", " << Inputs[i];
  }
  cout << ") = " << fixed << setprecision(precision) << Result << " (expect "
       << Expect - pow(10, -precision) << " ~ " << Expect + pow(10, -precision)
       << ")";
  cout.unsetf(ios::fixed);
  check(abs(Result - Expect) < pow(10, -precision));
}

void checkResult(const string &FuncName, const vector<__hip_bfloat16> &Inputs,
                 const __hip_bfloat16 &Expect, const float &Result,
                 const int precision) {
  vector<float> FInputs;
  for (const auto &it : Inputs) {
    FInputs.push_back(__bfloat162float(it));
  }
  float FExpect{__bfloat162float(Expect)};
  checkResult(FuncName, FInputs, FExpect, Result, precision);
}

// Bfloat16 Arithmetic Functions

__global__ void habs(float *const Result, __hip_bfloat16 Input1) {
  *Result = __habs(Input1);
}

void testHabsCases(const vector<pair<__hip_bfloat16, bf16i_pair>> &TestCases) {
  float *Result;
  hipMallocManaged(&Result, sizeof(*Result));
  for (const auto &TestCase : TestCases) {
    habs<<<1, 1>>>(Result, TestCase.first);
    hipDeviceSynchronize();
    checkResult("__habs", {TestCase.first}, TestCase.second.first, *Result,
                TestCase.second.second);
  }
}

int main() {
  testHabsCases({
      {{-0.3}, {0.30078125, 16}},
      {{0.3}, {0.30078125, 16}},
      {{0.5}, {0.5, 16}},
      {{0.4}, {0.400390625, 16}},
      {{6}, {6, 15}},
  });
  cout << "passed " << passed << "/" << passed + failed << " cases!" << endl;
  if (failed) {
    cout << "failed!" << endl;
  }
  return failed;
}
