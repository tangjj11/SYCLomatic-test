
#include <hip/hip_runtime.h>
// ====------ Image_api_test4.cu---------- *- CUDA -* ----===////
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//
// ===----------------------------------------------------------------------===//

// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test4_out

// CHECK: 2
// TEST_FEATURE: Image_image_channel_data_type

int main() {
  hipChannelFormatKind a;
  return 0;
}